#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"


extern float toBW(int bytes, float sec);


/* Helper function to round up to a power of 2.
 */
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void upsweep_kernel(int* data, int n, int stride) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int index = (i + 1) * stride * 2 - 1;
    
    if (index < n && index >= stride) {
        data[index] += data[index - stride];
    }
}

__global__ void downsweep_kernel(int* data, int n, int stride) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int index = (i + 1) * stride * 2 - 1;
    
    if (index < n && index >= stride) {
        int temp = data[index - stride];
        data[index - stride] = data[index];
        data[index] += temp;
    }
}

void exclusive_scan(int* device_data, int length)
{
    /* TODO
     * Fill in this function with your exclusive scan implementation.
     * You are passed the locations of the data in device memory
     * The data are initialized to the inputs.  Your code should
     * do an in-place scan, generating the results in the same array.
     * This is host code -- you will need to declare one or more CUDA
     * kernels (with the __global__ decorator) in order to actually run code
     * in parallel on the GPU.
     * Note you are given the real length of the array, but may assume that
     * both the data array is sized to accommodate the next
     * power of 2 larger than the input.
     */

    int n = nextPow2(length);
    
    for (int stride = 1; stride < n; stride *= 2) {
        int num_threads = n / (stride * 2);
        if (num_threads > 0) {
            int blocks = (num_threads + 255) / 256;
            if (blocks > 0) {
                upsweep_kernel<<<blocks, 256>>>(device_data, n, stride);
            }
        }
    }
    hipDeviceSynchronize();
    
    if (n > 0) {
        hipMemset(&device_data[n-1], 0, sizeof(int));
    }
    
    for (int stride = n/2; stride > 0; stride /= 2) {
        int num_threads = n / (stride * 2);
        if (num_threads > 0) {
            int blocks = (num_threads + 255) / 256;
            if (blocks > 0) {
                downsweep_kernel<<<blocks, 256>>>(device_data, n, stride);
            }
        }
    }
    hipDeviceSynchronize();
}

/* This function is a wrapper around the code you will write - it copies the
 * input to the GPU and times the invocation of the exclusive_scan() function
 * above. You should not modify it.
 */
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_data;
    // We round the array size up to a power of 2, but elements after
    // the end of the original input are left uninitialized and not checked
    // for correctness.
    // You may have an easier time in your implementation if you assume the
    // array's length is a power of 2, but this will result in extra work on
    // non-power-of-2 inputs.
    int rounded_length = nextPow2(end - inarray);
    hipMalloc((void **)&device_data, sizeof(int) * rounded_length);

    hipMemcpy(device_data, inarray, (end - inarray) * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_data, end - inarray);

    // Wait for any work left over to be completed.
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;

    hipMemcpy(resultarray, device_data, (end - inarray) * sizeof(int),
               hipMemcpyDeviceToHost);
    return overallDuration;
}

/* Wrapper around the Thrust library's exclusive scan function
 * As above, copies the input onto the GPU and times only the execution
 * of the scan itself.
 * You are not expected to produce competitive performance to the
 * Thrust version.
 */
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

    hipMemcpy(d_input.get(), inarray, length * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int),
               hipMemcpyDeviceToHost);
    thrust::device_free(d_input);
    thrust::device_free(d_output);
    double overallDuration = endTime - startTime;
    return overallDuration;
}


__global__ void mark_peaks_kernel(int* input, int* flags, int length) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < length) {
        if (i > 0 && i < length - 1) {
            if (input[i] > input[i - 1] && input[i] > input[i + 1]) {
                flags[i] = 1;
            } else {
                flags[i] = 0;
            }
        } else {
            flags[i] = 0;
        }
    }
}

__global__ void gather_indices_kernel(int* flags, int* scanned_flags, int* output, int length) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < length && flags[i] == 1) {
        output[scanned_flags[i]] = i;
    }
}

int find_peaks(int *device_input, int length, int *device_output) {
    /* TODO:
     * Finds all elements in the list that are greater than the elements before and after,
     * storing the index of the element into device_result.
     * Returns the number of peak elements found.
     * By definition, neither element 0 nor element length-1 is a peak.
     *
     * Your task is to implement this function. You will probably want to
     * make use of one or more calls to exclusive_scan(), as well as
     * additional CUDA kernel launches.
     * Note: As in the scan code, we ensure that allocated arrays are a power
     * of 2 in size, so you can use your exclusive_scan function with them if
     * it requires that. However, you must ensure that the results of
     * find_peaks are correct given the original length.
     */

    if (length <= 2) {
        return 0;
    }
    
    int rounded_length = nextPow2(length);
    int *device_flags;
    int *device_scanned_flags;
    
    hipMalloc((void**)&device_flags, rounded_length * sizeof(int));
    hipMalloc((void**)&device_scanned_flags, rounded_length * sizeof(int));
    
    hipMemset(device_flags, 0, rounded_length * sizeof(int));
    hipMemset(device_scanned_flags, 0, rounded_length * sizeof(int));
    
    int blocks = (length + 255) / 256;
    if (blocks > 0) {
        mark_peaks_kernel<<<blocks, 256>>>(device_input, device_flags, length);
        hipDeviceSynchronize();
    }
    
    hipMemcpy(device_scanned_flags, device_flags, length * sizeof(int), 
               hipMemcpyDeviceToDevice);
    
    exclusive_scan(device_scanned_flags, length);
    
    int total_peaks = 0;
    if (length > 0) {
        int last_flag, last_scanned;
        hipMemcpy(&last_flag, &device_flags[length-1], sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&last_scanned, &device_scanned_flags[length-1], sizeof(int), hipMemcpyDeviceToHost);
        total_peaks = last_scanned + last_flag;
    }
    
    if (total_peaks > 0 && blocks > 0) {
        gather_indices_kernel<<<blocks, 256>>>(device_flags, device_scanned_flags, device_output, length);
        hipDeviceSynchronize();
    }
    
    hipFree(device_flags);
    hipFree(device_scanned_flags);
    
    return total_peaks;
}


/* Timing wrapper around find_peaks. You should not modify this function.
 */
double cudaFindPeaks(int *input, int length, int *output, int *output_length) {
    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int),
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    int result = find_peaks(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    *output_length = result;

    hipMemcpy(output, device_output, length * sizeof(int),
               hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    return endTime - startTime;
}


void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
